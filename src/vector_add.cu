#include "hip/hip_runtime.h"
#include "vector_add.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

__global__ void vectorAddKernel(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

void vectorAdd(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel (256 threads per block)
    int threadsPerBlock = 256;
    int blocksPerGrid = static_cast<int>(N / threadsPerBlock); //(N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
